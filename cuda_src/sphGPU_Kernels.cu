#include "hip/hip_runtime.h"
#include "../cuda_inc/sphGPU_Kernels.cuh"


#include "../cuda_inc/vec_ops.cuh"
#include "../cuda_inc/smoothingKernel.cuh"


#include <hip/hip_math_constants.h>
#include <stdio.h>
#include <math.h>
#include <float.h>



__global__ void sphGPU_Kernels::ParticleHash_Kernel(uint *hash,
                                                    uint *cellOcc,
                                                    const float3 *particles,
                                                    const uint N,
                                                    const uint gridRes,
                                                    const float cellWidth)
{
    uint idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Sanity check
    if (idx >= N)
    {
        return;
    }

    float gridDim = gridRes * cellWidth;
    float invGridDim = 1.0f / gridDim;
    float3 particle = particles[idx];
    uint hashID;

    // Get normalised particle positions [0-1]
    float normX = (particle.x + (0.5f * gridDim)) * invGridDim;
    float normY = (particle.y + (0.5f * gridDim)) * invGridDim;
    float normZ = (particle.z + (0.5f * gridDim)) * invGridDim;


    // Get hash values for x, y, z
    uint hashX = floor(normX * gridRes);
    uint hashY = floor(normY * gridRes);
    uint hashZ = floor(normZ * gridRes);

    hashX = (hashX >= gridRes) ? gridRes-1 : hashX;
    hashY = (hashY >= gridRes) ? gridRes-1 : hashY;
    hashZ = (hashZ >= gridRes) ? gridRes-1 : hashZ;

    hashID = hashX + (hashY * gridRes) + (hashZ * gridRes * gridRes);

    if(hashID >= gridRes * gridRes * gridRes)
    {
        printf("Hash out of bounds\n");
        printf("%u, %u, %u\n", hashX, hashY, hashZ);
    }

    // Update hash id for this particle
    hash[idx] = hashID;


    // Update cell occupancy for the cell
    atomicAdd(&cellOcc[hashID], 1u);


}

__global__ void sphGPU_Kernels::ComputeVolume_kernel(float *volume,
                                     const uint *cellOcc,
                                     const uint *cellPartIdx,
                                     const float3 *particles,
                                     const uint numPoints,
                                     const float smoothingLength)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;

    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float accVolume = 0.0f;
        float3 thisParticle = particles[thisParticleGlobalIdx];

        uint numNeighCells = 0;
        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {
                    numNeighCells++;
                    neighCellIdx = thisCellIdx + x + (y*gridDim.x) + (z*gridDim.x*gridDim.y);

                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];
                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        float3 neighParticle = particles[neighParticleGlobalIdx];

                        accVolume += fabs(Poly6Kernel_Kernel(length(thisParticle - neighParticle), smoothingLength));
                    }
                }
            }
        }

        if(isnan(accVolume) || fabs(accVolume) < FLT_EPSILON)
        {
            volume[thisParticleGlobalIdx] = 1.0f;
        }
        else
        {
            volume[thisParticleGlobalIdx] = accVolume;
        }
    }
}

__global__ void sphGPU_Kernels::ComputeDensity_kernel(float *density,
                                                      const float *mass,
                                                      const uint *cellOcc,
                                                      const uint *cellPartIdx,
                                                      const float3 *particles,
                                                      const uint numPoints,
                                                      const float smoothingLength,
                                                      const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;



    if((thisParticleGlobalIdx < numPoints) && (threadIdx.x < cellOcc[thisCellIdx]) && (thisCellIdx < gridDim.x * gridDim.y * gridDim.z))
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float accDensity = 0.0f;
        float thisDensity = 0.0f;
        float3 thisParticle = particles[thisParticleGlobalIdx];

        unsigned int numNeighs = 0;
        uint numNeighCells = 0;
        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {
                    numNeighCells++;
                    neighCellIdx = thisCellIdx + x + (y*gridDim.x) + (z*gridDim.x*gridDim.y);

                    // Get density contribution from other fluid particles
                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];
                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;

                        float3 neighParticle = particles[neighParticleGlobalIdx];

                        thisDensity = mass[neighParticleGlobalIdx] * fabs(Poly6Kernel_Kernel(length(thisParticle - neighParticle), smoothingLength));

                        accDensity += thisDensity;

                        numNeighs++;
                    }
                }
            }
        }

        if(isnan(accDensity))
        {
            printf("nan density \n");

            if(!accumulate)
            {
                density[thisParticleGlobalIdx] = 0.0f;
            }
        }
        else
        {
            if(accumulate)
            {
                atomicAdd(&density[thisParticleGlobalIdx], accDensity);
            }
            else
            {
                density[thisParticleGlobalIdx] = accDensity;
            }
        }


    }

}



__global__ void sphGPU_Kernels::ComputeDensityFluidRigid_kernel(const uint numPoints,
                                                                const float fluidRestDensity,
                                                                float *fluidDensity,
                                                                const uint *fluidCellOcc,
                                                                const uint *fluidCellPartIdx,
                                                                const float3 *fluidPos,
                                                                float *rigidVolume,
                                                                const uint *rigidCellOcc,
                                                                const uint *rigidCellPartIdx,
                                                                const float3 *rigidPos,
                                                                const float smoothingLength,
                                                                const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = fluidCellPartIdx[thisCellIdx] + threadIdx.x;



    if((thisParticleGlobalIdx < numPoints) && (threadIdx.x < fluidCellOcc[thisCellIdx]) && (thisCellIdx < gridDim.x * gridDim.y * gridDim.z))
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float accDensity = 0.0f;
        float thisDensity = 0.0f;
        float3 thisParticle = fluidPos[thisParticleGlobalIdx];

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {
                    neighCellIdx = thisCellIdx + x + (y*gridDim.x) + (z*gridDim.x*gridDim.y);

                    // Get density contribution from other fluid particles
                    neighCellOcc = rigidCellOcc[neighCellIdx];
                    neighCellPartIdx = rigidCellPartIdx[neighCellIdx];
                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;

                        float3 neighParticle = rigidPos[neighParticleGlobalIdx];

                        thisDensity = fluidRestDensity * rigidVolume[neighParticleGlobalIdx] * fabs(Poly6Kernel_Kernel(length(thisParticle - neighParticle), smoothingLength));

                        accDensity += thisDensity;
                    }
                }
            }
        }

        if(isnan(accDensity))
        {
            printf("nan density \n");

            if(!accumulate)
            {
                fluidDensity[thisParticleGlobalIdx] = 0.0f;
            }
        }
        else
        {
            if(accumulate)
            {
                atomicAdd(&fluidDensity[thisParticleGlobalIdx], accDensity);
            }
            else
            {
                fluidDensity[thisParticleGlobalIdx] = accDensity;
            }
        }

    } // end if valid point
}



__global__ void sphGPU_Kernels::ComputeDensityFluidFluid_kernel(const uint numPoints,
                                                                float *fluidDensity,
                                                                const uint *fluidCellOcc,
                                                                const uint *fluidCellPartIdx,
                                                                const float3 *fluidPos,
                                                                const uint *otherFluidCellOcc,
                                                                const uint *otherFluidCellPartIdx,
                                                                float *otherFluidMass,
                                                                const float3 *otherFluidPos,
                                                                const float smoothingLength,
                                                                const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = fluidCellPartIdx[thisCellIdx] + threadIdx.x;



    if((thisParticleGlobalIdx < numPoints) && (threadIdx.x < fluidCellOcc[thisCellIdx]) && (thisCellIdx < gridDim.x * gridDim.y * gridDim.z))
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float accDensity = 0.0f;
        float thisDensity = 0.0f;
        float3 thisParticle = fluidPos[thisParticleGlobalIdx];

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {
                    neighCellIdx = thisCellIdx + x + (y*gridDim.x) + (z*gridDim.x*gridDim.y);

                    // Get density contribution from other fluid particles
                    neighCellOcc = otherFluidCellOcc[neighCellIdx];
                    neighCellPartIdx = otherFluidCellPartIdx[neighCellIdx];
                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;

                        float3 neighParticle = otherFluidPos[neighParticleGlobalIdx];

                        thisDensity = otherFluidMass[neighParticleGlobalIdx] * fabs(Poly6Kernel_Kernel(length(thisParticle - neighParticle), smoothingLength));

                        accDensity += thisDensity;
                    }
                }
            }
        }

        if(isnan(accDensity))
        {
            printf("nan density \n");

            if(!accumulate)
            {
                fluidDensity[thisParticleGlobalIdx] = 0.0f;
            }
        }
        else
        {
            if(accumulate)
            {
                atomicAdd(&fluidDensity[thisParticleGlobalIdx], accDensity);
            }
            else
            {
                fluidDensity[thisParticleGlobalIdx] = accDensity;
            }
        }

    } // end if valid point
}






__global__ void sphGPU_Kernels::ComputePressure_kernel(float *pressure,
                                                       float *density,
                                                       const float restDensity,
                                                       const float gasConstant,
                                                       const uint *cellOcc,
                                                       const uint *cellPartIdx,
                                                       const uint numPoints)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;



    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        //float beta = 0.35;
        //float gamma = 7.0f;
        //float accPressure = beta * (pow((accDensity/restDensity), gamma)-1.0f);
        //float accPressure = gasConstant * ((accDensity/restDensity) - 1.0f);

        float accPressure = gasConstant * (density[thisParticleGlobalIdx] - restDensity);

        if(isnan(accPressure))
        {
            printf("nan pressure \n");
            pressure[thisParticleGlobalIdx] = 0.0f;
        }
        else
        {
            pressure[thisParticleGlobalIdx] = accPressure;
        }

    }

}

__global__ void sphGPU_Kernels::ComputePressureForce_kernel(float3 *pressureForce,
                                                            const float *pressure,
                                                            const float *density,
                                                            const float *mass,
                                                            const float3 *particles,
                                                            const uint *cellOcc,
                                                            const uint *cellPartIdx,
                                                            const uint numPoints,
                                                            const float smoothingLength,
                                                            const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;


    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {

        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float3 accPressureForce = make_float3(0.0f, 0.0f, 0.0f);


        float thisPressure = pressure[thisParticleGlobalIdx];
        float3 thisParticle = particles[thisParticleGlobalIdx];

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        if(neighParticleGlobalIdx != thisParticleGlobalIdx)
                        {
                            float3 neighParticle = particles[neighParticleGlobalIdx];
                            float neighPressure = pressure[neighParticleGlobalIdx];
                            float neighDensity = density[neighParticleGlobalIdx];
                            float neighMass = mass[neighParticleGlobalIdx];

                            float pressOverDens = (fabs(neighDensity)<FLT_EPSILON ? 0.0f: (thisPressure + neighPressure) / (2.0f* neighDensity));

                            accPressureForce = accPressureForce + (neighMass * pressOverDens * SpikyKernelGradientV_Kernel(thisParticle, neighParticle, smoothingLength));
                        }
                    }
                }
            }
        }


        if(accumulate)
        {
            pressureForce[thisParticleGlobalIdx] = -1.0f * accPressureForce;
        }
        else
        {
            pressureForce[thisParticleGlobalIdx] = pressureForce[thisParticleGlobalIdx] + (-1.0f * accPressureForce);
        }
    }
}

__global__ void sphGPU_Kernels::ComputeViscousForce_kernel(float3 *viscForce,
                                                           const float viscCoeff,
                                                           const float3 *velocity,
                                                           const float *density,
                                                           const float *mass,
                                                           const float3 *position,
                                                           const uint *cellOcc,
                                                           const uint *cellPartIdx,
                                                           const uint numPoints,
                                                           const float smoothingLength)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;


    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float3 accViscForce = make_float3(0.0f, 0.0f, 0.0f);


        float3 thisPos = position[thisParticleGlobalIdx];
        float3 thisVel = velocity[thisParticleGlobalIdx];

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        if(neighParticleGlobalIdx == thisParticleGlobalIdx){continue;}

                        float3 neighPos = position[neighParticleGlobalIdx];
                        float3 neighVel = velocity[neighParticleGlobalIdx];
                        float neighDensity = density[neighParticleGlobalIdx];
                        float neighMass = mass[neighParticleGlobalIdx];
                        float neighMassOverDen = ( (fabs(neighDensity)<FLT_EPSILON) ? 0.0f : neighMass / neighDensity );

                        accViscForce = accViscForce + ( neighMassOverDen * (neighVel - thisVel) * Poly6Laplacian_Kernel(length(thisPos - neighPos), smoothingLength) );
                    }
                }
            }
        }

        viscForce[thisParticleGlobalIdx] = -1.0f * viscCoeff * accViscForce;
    }
}


__global__ void sphGPU_Kernels::ComputeSurfaceTensionForce_kernel(float3 *surfaceTensionForce,
                                                                  const float surfaceTension,
                                                                  const float surfaceThreshold,
                                                                  /*const*/ float *density,
                                                                  const float *mass,
                                                                  const float3 *position,
                                                                  const uint *cellOcc,
                                                                  const uint *cellPartIdx,
                                                                  const uint numPoints,
                                                                  const float smoothingLength)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;


    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;


        float3 thisPos = position[thisParticleGlobalIdx];
        float3 accColourFieldGrad = make_float3(0.0f, 0.0f, 0.0f);
        float accCurvature = 0.0f;

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        if(neighParticleGlobalIdx == thisParticleGlobalIdx){continue;}

                        float3 neighPos = position[neighParticleGlobalIdx];
                        float neighDensity = density[neighParticleGlobalIdx];
                        float neighMass = mass[neighParticleGlobalIdx];
                        float neighMassOverDen = ( (fabs(neighDensity)<FLT_EPSILON) ? 0.0f : neighMass / neighDensity );

                        accColourFieldGrad = accColourFieldGrad + ( neighMassOverDen * SpikyKernelGradientV_Kernel(thisPos, neighPos, smoothingLength) );
                        accCurvature = accCurvature + (neighMassOverDen * -Poly6Laplacian_Kernel(length(thisPos - neighPos), smoothingLength));

                    }
                }
            }
        }

        float colourFieldGradMag = length(accColourFieldGrad);
        if( colourFieldGradMag > surfaceThreshold )
        {
            accCurvature /= colourFieldGradMag;
            surfaceTensionForce[thisParticleGlobalIdx] = (-surfaceTension * accCurvature * accColourFieldGrad);
        }
        else
        {
            surfaceTensionForce[thisParticleGlobalIdx] = make_float3(0.0f, 0.0f, 0.0f);
        }
    }
}

__global__ void sphGPU_Kernels::ComputeForces_kernel(float3 *force,
                                                     const float3 *externalForce,
                                                     const float3 *pressureForce,
                                                     const float3 *viscousForce,
                                                     const float3 *surfaceTensionForce,
                                                     const float *mass,
                                                     const float3 *particles,
                                                     const float3 *velocities,
                                                     const uint *cellOcc,
                                                     const uint *cellPartIdx,
                                                     const uint numPoints,
                                                     const float smoothingLength)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;

    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        // re-initialise forces to zero
        float3 accForce = make_float3(0.0f, 0.0f, 0.0f);

        // Add external force
        float3 extForce = externalForce[thisCellIdx];
        if(isnan(extForce.x) || isnan(extForce.y) || isnan(extForce.z))
        {
            printf("nan external force\n");
        }
        else
        {
            accForce = accForce + extForce;
        }


        // Add pressure force
        float3 pressForce = pressureForce[thisParticleGlobalIdx];
        if(isnan(pressForce.x) || isnan(pressForce.y) || isnan(pressForce.z))
        {
            printf("nan pressure force\n");
        }
        else
        {
            accForce = accForce + pressForce;
        }

        // Add Viscous force
        float3 viscForce = viscousForce[thisParticleGlobalIdx];
        if(isnan(viscForce.x) || isnan(viscForce.y) || isnan(viscForce.z))
        {
            printf("nan visc force\n");
        }
        else
        {
            accForce = accForce + viscForce;
        }

        // Add surface tension force
        float3 surfTenForce = surfaceTensionForce[thisParticleGlobalIdx];
        if(isnan(surfTenForce.x) || isnan(surfTenForce.y) || isnan(surfTenForce.z))
        {
            printf("nan surfTen force\n");
        }
        else
        {
            //printf("%f, %f, %f\n",surfTenForce.x, surfTenForce.y,surfTenForce.z);
            accForce = accForce + surfTenForce;
        }


        // Work out acceleration from force
        float3 acceleration = accForce / mass[thisParticleGlobalIdx];

        // Add gravity acceleration
        acceleration = acceleration + make_float3(0.0f, -9.81f, 0.0f);

        // Set particle force
        force[thisParticleGlobalIdx] = acceleration;
    }
}

__global__ void sphGPU_Kernels::Integrate_kernel(float3 *force,
                                                 float3 *particles,
                                                 float3 *velocities,
                                                 const float _dt,
                                                 const uint numPoints)
{
    uint idx = threadIdx.x + (blockIdx.x * blockDim.x);

    if(idx < numPoints)
    {
        //---------------------------------------------------------
        // Good old instable Euler integration - ONLY FOR TESTING
        float3 oldPos = particles[idx];
        float3 oldVel = velocities[idx];

        float3 newVel = oldVel + (_dt * force[idx]);
        float3 newPos = oldPos + (_dt * newVel);

        //---------------------------------------------------------
        // Verlet/Leapfrog integration
//        float3 newPos = oldPos + (oldVel * _dt) + (0.5f * force[idx] * _dt * _dt);
//        float3 newVel = oldVel + (0.5 * (force[idx] + force[idx]) * _dt);

        //---------------------------------------------------------
        // TODO:
        // Verlet integration
        // RK4 integration

        //---------------------------------------------------------
        // Error checking and setting new values

        if(isnan(newVel.x) || isnan(newVel.y) || isnan(newVel.z))
        {
            printf("nan vel\n");
        }
        else
        {
            velocities[idx] = newVel;
        }

        if(isnan(newPos.x) || isnan(newPos.y) || isnan(newPos.z))
        {
            printf("nan pos\n");
        }
        else
        {
            particles[idx] = newPos;
        }
    }
}

__global__ void sphGPU_Kernels::HandleBoundaries_Kernel(float3 *particles,
                                                        float3 *velocities,
                                                        const float boundary,
                                                        const uint numPoints)
{
    uint idx = threadIdx.x + (blockIdx.x * blockDim.x);

    if(idx < numPoints)
    {

        float3 pos = particles[idx];
        float3 vel = velocities[idx];

        float boundaryDamper = 0.4f;

        if(pos.x < -boundary)
        {
           pos.x = -boundary  + fabs(fabs(pos.x) - boundary);
           vel = make_float3(boundaryDamper*fabs(vel.x),vel.y,vel.z);
        }
        if(pos.x > boundary)
        {
           pos.x = boundary - fabs(fabs(pos.x) - boundary);
           vel = make_float3(-boundaryDamper*fabs(vel.x),vel.y,vel.z);
        }

        if(pos.y < -boundary)
        {
           pos.y = -boundary + fabs(fabs(pos.y) - boundary);
           vel = make_float3(vel.x,boundaryDamper*fabs(vel.y),vel.z);
        }
        if(pos.y > boundary)
        {
           pos.y = boundary - fabs(fabs(pos.y) - boundary);
           vel = make_float3(vel.x,-boundaryDamper*fabs(vel.y),vel.z);
        }

        if(pos.z < -boundary)
        {
           pos.z = -boundary + fabs(fabs(pos.z) - boundary);
           vel = make_float3(vel.x,vel.y,boundaryDamper*fabs(vel.z));
        }
        if(pos.z > boundary)
        {
           pos.z = boundary - fabs(fabs(pos.z) - boundary);
           vel = make_float3(vel.x,vel.y,-boundaryDamper*fabs(vel.z));
        }

        particles[idx] = pos;
        velocities[idx] = vel;
    }
}

__global__ void sphGPU_Kernels::InitParticleAsCube_Kernel(float3 *particles,
                                                          float3 *velocities,
                                                          float *densities,
                                                          const float restDensity,
                                                          const uint numParticles,
                                                          const uint numPartsPerAxis,
                                                          const float scale)
{

    uint x = threadIdx.x + (blockIdx.x * blockDim.x);
    uint y = threadIdx.y + (blockIdx.y * blockDim.y);
    uint z = threadIdx.z + (blockIdx.z * blockDim.z);
    uint idx = x + (y * numPartsPerAxis) + (z * numPartsPerAxis * numPartsPerAxis);

    if(x >= numPartsPerAxis || y >= numPartsPerAxis || z >= numPartsPerAxis || idx >= numParticles)
    {
        return;
    }

    float posX = scale * (x - (0.5f * numPartsPerAxis));
    float posY = scale * (y - (0.5f * numPartsPerAxis));
    float posZ = scale * (z - (0.5f * numPartsPerAxis));

    particles[idx] = make_float3(posX, posY, posZ);
    velocities[idx] = make_float3(0.0f, 0.0f, 0.0f);
    densities[idx] = restDensity;
}



