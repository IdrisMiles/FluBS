#include "hip/hip_runtime.h"
#include "../cuda_inc/sphGPU_Kernels.cuh"


#include "../cuda_inc/vec_ops.cuh"
#include "../cuda_inc/smoothingKernel.cuh"

#include <functional>

#include <hip/hip_math_constants.h>
#include <stdio.h>
#include <math.h>
#include <float.h>


//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ParticleHash_Kernel(uint *hash,
                                                    uint *cellOcc,
                                                    const float3 *particles,
                                                    const uint N,
                                                    const uint gridRes,
                                                    const float cellWidth)
{
    uint idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Sanity check
    if (idx >= N)
    {
        return;
    }

    float gridDim = gridRes * cellWidth;
    float invGridDim = 1.0f / gridDim;
    float3 particle = particles[idx];
    uint hashID;

    // Get normalised particle positions [0-1]
    float normX = (particle.x + (0.5f * gridDim)) * invGridDim;
    float normY = (particle.y + (0.5f * gridDim)) * invGridDim;
    float normZ = (particle.z + (0.5f * gridDim)) * invGridDim;


    // Get hash values for x, y, z
    uint hashX = floor(normX * gridRes);
    uint hashY = floor(normY * gridRes);
    uint hashZ = floor(normZ * gridRes);

    hashX = (hashX >= gridRes) ? gridRes-1 : hashX;
    hashY = (hashY >= gridRes) ? gridRes-1 : hashY;
    hashZ = (hashZ >= gridRes) ? gridRes-1 : hashZ;

    hashX = (hashX < 0) ? 0 : hashX;
    hashY = (hashY < 0) ? 0 : hashY;
    hashZ = (hashZ < 0) ? 0 : hashZ;

    hashID = hashX + (hashY * gridRes) + (hashZ * gridRes * gridRes);

    if(hashID >= gridRes * gridRes * gridRes || hashID < 0)
    {
        printf("Hash out of bounds\n");
        printf("%u, %u, %u\n", hashX, hashY, hashZ);
    }

    // Update hash id for this particle
    hash[idx] = hashID;


    // Update cell occupancy for the cell
    atomicAdd(&cellOcc[hashID], 1u);


}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputeVolume_kernel(float *volume,
                                     const uint *cellOcc,
                                     const uint *cellPartIdx,
                                     const float3 *particles,
                                     const uint numPoints,
                                     const float smoothingLength)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;

    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float accVolume = 0.0f;
        float3 thisParticle = particles[thisParticleGlobalIdx];

        uint numNeighCells = 0;
        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {
                    numNeighCells++;
                    neighCellIdx = thisCellIdx + x + (y*gridDim.x) + (z*gridDim.x*gridDim.y);

                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];
                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        float3 neighParticle = particles[neighParticleGlobalIdx];

                        accVolume += fabs(Poly6Kernel_Kernel(length(thisParticle - neighParticle), smoothingLength));
                    }
                }
            }
        }

        accVolume = 1.0f / accVolume;

        if(isnan(accVolume) || fabs(accVolume) < FLT_EPSILON)
        {
            volume[thisParticleGlobalIdx] = 1.0f;
        }
        else
        {
            volume[thisParticleGlobalIdx] = 10.0f*accVolume;
        }
    }
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputeDensity_kernel(float *density,
                                                      const float mass,
                                                      const uint *cellOcc,
                                                      const uint *cellPartIdx,
                                                      const float3 *particles,
                                                      const uint numPoints,
                                                      const float smoothingLength,
                                                      const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;



    if((thisParticleGlobalIdx < numPoints) && (threadIdx.x < cellOcc[thisCellIdx]) && (thisCellIdx < gridDim.x * gridDim.y * gridDim.z))
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float accDensity = 0.0f;
        float thisDensity = 0.0f;
        float thisMass = mass;
        float3 thisParticle = particles[thisParticleGlobalIdx];

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {
                    neighCellIdx = thisCellIdx + x + (y*gridDim.x) + (z*gridDim.x*gridDim.y);

                    // Get density contribution from other fluid particles
                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];
                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;

                        float3 neighParticle = particles[neighParticleGlobalIdx];

                        thisDensity = thisMass * Poly6Kernel_Kernel(length(thisParticle - neighParticle), smoothingLength);

                        accDensity += thisDensity;
                    }
                }
            }
        }

        if(isnan(accDensity))
        {
            printf("nan density \n");

            if(!accumulate)
            {
                density[thisParticleGlobalIdx] = 0.0f;
            }
        }
        else
        {
            if(accumulate)
            {
                atomicAdd(&density[thisParticleGlobalIdx], accDensity);
            }
            else
            {
                density[thisParticleGlobalIdx] = accDensity;
            }
        }


    }

}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputeDensityFluidRigid_kernel(const uint numPoints,
                                                                const float fluidRestDensity,
                                                                float *fluidDensity,
                                                                const uint *fluidCellOcc,
                                                                const uint *fluidCellPartIdx,
                                                                const float3 *fluidPos,
                                                                float *rigidVolume,
                                                                const uint *rigidCellOcc,
                                                                const uint *rigidCellPartIdx,
                                                                const float3 *rigidPos,
                                                                const float smoothingLength,
                                                                const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = fluidCellPartIdx[thisCellIdx] + threadIdx.x;



    if((thisParticleGlobalIdx < numPoints) && (threadIdx.x < fluidCellOcc[thisCellIdx]) && (thisCellIdx < gridDim.x * gridDim.y * gridDim.z))
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float accDensity = 0.0f;
        float thisDensity = 0.0f;
        float3 thisParticle = fluidPos[thisParticleGlobalIdx];

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {
                    neighCellIdx = thisCellIdx + x + (y*gridDim.x) + (z*gridDim.x*gridDim.y);

                    // Get density contribution from other fluid particles
                    neighCellOcc = rigidCellOcc[neighCellIdx];
                    neighCellPartIdx = rigidCellPartIdx[neighCellIdx];
                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;

                        float3 neighParticle = rigidPos[neighParticleGlobalIdx];

                        thisDensity = fluidRestDensity * rigidVolume[neighParticleGlobalIdx] * Poly6Kernel_Kernel(length(thisParticle - neighParticle), smoothingLength);

                        accDensity += (thisDensity);
                    }
                }
            }
        }

        if(isnan(accDensity))
        {
            printf("nan density \n");

            if(!accumulate)
            {
                fluidDensity[thisParticleGlobalIdx] = 0.0f;
            }
        }
        else
        {
            if(accumulate)
            {
                atomicAdd(&fluidDensity[thisParticleGlobalIdx], accDensity);
            }
            else
            {
                fluidDensity[thisParticleGlobalIdx] = accDensity;
            }
        }

    } // end if valid point
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputeDensityFluidFluid_kernel(const uint numPoints,
                                                                float *fluidDensity,
                                                                const uint *fluidCellOcc,
                                                                const uint *fluidCellPartIdx,
                                                                const float3 *fluidPos,
                                                                const uint *otherFluidCellOcc,
                                                                const uint *otherFluidCellPartIdx,
                                                                const float otherFluidMass,
                                                                const float3 *otherFluidPos,
                                                                const float smoothingLength,
                                                                const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = fluidCellPartIdx[thisCellIdx] + threadIdx.x;



    if((thisParticleGlobalIdx < numPoints) && (threadIdx.x < fluidCellOcc[thisCellIdx]) && (thisCellIdx < gridDim.x * gridDim.y * gridDim.z))
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float accDensity = 0.0f;
        float thisDensity = 0.0f;
        float3 thisParticle = fluidPos[thisParticleGlobalIdx];

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {
                    neighCellIdx = thisCellIdx + x + (y*gridDim.x) + (z*gridDim.x*gridDim.y);

                    // Get density contribution from other fluid particles
                    neighCellOcc = otherFluidCellOcc[neighCellIdx];
                    neighCellPartIdx = otherFluidCellPartIdx[neighCellIdx];
                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;

                        float3 neighParticle = otherFluidPos[neighParticleGlobalIdx];

                        thisDensity = otherFluidMass * Poly6Kernel_Kernel(length(thisParticle - neighParticle), smoothingLength);

                        accDensity += thisDensity;
                    }
                }
            }
        }

        if(isnan(accDensity))
        {
            printf("nan density \n");

            if(!accumulate)
            {
                fluidDensity[thisParticleGlobalIdx] = 0.0f;
            }
        }
        else
        {
            if(accumulate)
            {
                atomicAdd(&fluidDensity[thisParticleGlobalIdx], accDensity);
            }
            else
            {
                fluidDensity[thisParticleGlobalIdx] = accDensity;
            }
        }

    } // end if valid point
}



//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputePressure_kernel(float *pressure,
                                                       float *density,
                                                       const float restDensity,
                                                       const float gasConstant,
                                                       const uint *cellOcc,
                                                       const uint *cellPartIdx,
                                                       const uint numPoints)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;



    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        //float beta = 0.35;
        //float gamma = 7.0f;
        //float accPressure = beta * (pow((accDensity/restDensity), gamma)-1.0f);
        //float accPressure = gasConstant * ((accDensity/restDensity) - 1.0f);

//        float k = 50.0f;
        float gamma = 7.0f;
        float accPressure = (gasConstant*restDensity / gamma) * (pow((density[thisParticleGlobalIdx]/restDensity), gamma) - 1.0f);

//        float accPressure = gasConstant * (density[thisParticleGlobalIdx] - restDensity);
//        float accPressure = gasConstant * ( (density[thisParticleGlobalIdx] / restDensity) - 1.0f);

        if(isnan(accPressure))
        {
            printf("nan pressure \n");
            pressure[thisParticleGlobalIdx] = 0.0f;
        }
        else
        {
            pressure[thisParticleGlobalIdx] = accPressure;
        }

    }

}

__global__ void sphGPU_Kernels::SamplePressure(const float3* samplePoints,
                                               float *pressure,
                                               const uint *cellOcc,
                                               const uint *cellPartIdx,
                                               const float3 *fluidPos,
                                               const float *fluidPressure,
                                               const float *fluidDensity,
                                               const float fluidParticleMass,
                                               const uint *fluidCellOcc,
                                               const uint *fluidCellPartIdx,
                                               const uint numPoints,
                                               const float smoothingLength)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = fluidCellPartIdx[thisCellIdx] + threadIdx.x;



    if((thisParticleGlobalIdx < numPoints) && (threadIdx.x < fluidCellOcc[thisCellIdx]) && (thisCellIdx < gridDim.x * gridDim.y * gridDim.z))
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float accPressue = 0.0f;
        float3 thisPos = samplePoints[thisParticleGlobalIdx];

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {
                    neighCellIdx = thisCellIdx + x + (y*gridDim.x) + (z*gridDim.x*gridDim.y);

                    neighCellOcc = fluidCellOcc[neighCellIdx];
                    neighCellPartIdx = fluidCellPartIdx[neighCellIdx];
                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;

                        float3 neighPos = fluidPos[neighParticleGlobalIdx];
                        float W = Poly6Kernel_Kernel(length(thisPos-neighPos), smoothingLength);
                        float invDen = 1.0f / fluidDensity[neighParticleGlobalIdx];
                        accPressue += invDen * W * fluidPressure[neighParticleGlobalIdx];
                    }
                }
            }
        }

        accPressue *= fluidParticleMass;

        pressure[thisParticleGlobalIdx] = accPressue;
    }
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputePressureForce_kernel(float3 *pressureForce,
                                                            const float *pressure,
                                                            const float *density,
                                                            const float mass,
                                                            const float3 *particles,
                                                            const uint *cellOcc,
                                                            const uint *cellPartIdx,
                                                            const uint numPoints,
                                                            const float smoothingLength,
                                                            const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;


    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {

        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float3 accPressureForce = make_float3(0.0f, 0.0f, 0.0f);


        float thisMass = mass;
        float thisDensity = density[thisParticleGlobalIdx];
        float thisPressure = pressure[thisParticleGlobalIdx];
        float3 thisParticle = particles[thisParticleGlobalIdx];

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        if(neighParticleGlobalIdx != thisParticleGlobalIdx)
                        {
                            float3 neighParticle = particles[neighParticleGlobalIdx];
                            float neighPressure = pressure[neighParticleGlobalIdx];
                            float neighDensity = density[neighParticleGlobalIdx];

//                            float pressOverDens = (fabs(neighDensity)<FLT_EPSILON ? 0.0f: (thisPressure + neighPressure) / (2.0f* neighDensity));

//                            accPressureForce = accPressureForce + (thisMass * pressOverDens * SpikyKernelGradientV_Kernel(thisParticle, neighParticle, smoothingLength));
//                            accPressureForce = accPressureForce + (thishMass * (thisPressure+neighPressure) / (neighDensity + neighDensity) * SpikyKernelGradientV_Kernel(thisParticle, neighParticle, smoothingLength));


                            accPressureForce = accPressureForce + ( ((thisPressure/(thisDensity*thisDensity)) + (neighPressure/(neighDensity*neighDensity))) * SpikyKernelGradientV_Kernel(thisParticle, neighParticle, smoothingLength) );
                        }
                    }
                }
            }
        }


        if(!accumulate)
        {
            pressureForce[thisParticleGlobalIdx] = -1.0f * thisMass * thisMass * accPressureForce;
//            pressureForce[thisParticleGlobalIdx] = -1.0f * accPressureForce;
        }
        else
        {
            pressureForce[thisParticleGlobalIdx] = pressureForce[thisParticleGlobalIdx] + (-1.0f * thisMass * thisMass * accPressureForce);
//            pressureForce[thisParticleGlobalIdx] = pressureForce[thisParticleGlobalIdx] + (-1.0f * accPressureForce);
        }
    }
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputePressureForceFluidFluid_kernel(float3 *pressureForce,
                                                                      const float *pressure,
                                                                      const float *density,
                                                                      const float mass,
                                                                      const float3 *particles,
                                                                      const uint *cellOcc,
                                                                      const uint *cellPartIdx,
                                                                      const float *fluidContribPressure,
                                                                      const float *fluidContribDensity,
                                                                      const float fluidContribMass,
                                                                      const float3 *fluidContribParticles,
                                                                      const uint *fluidContribCellOcc,
                                                                      const uint *fluidContribCellPartIdx,
                                                                      const uint numPoints,
                                                                      const float smoothingLength,
                                                                      const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;


    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {

        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float3 accPressureForce = make_float3(0.0f, 0.0f, 0.0f);


        float thisMass = mass;
        float thisDensity = density[thisParticleGlobalIdx];
        float thisPressure = pressure[thisParticleGlobalIdx];
        float3 thisParticle = particles[thisParticleGlobalIdx];
//        float neighMass = fluidContribMass;

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = fluidContribCellOcc[neighCellIdx];
                    neighCellPartIdx = fluidContribCellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        if(neighParticleGlobalIdx != thisParticleGlobalIdx)
                        {
                            float3 neighParticle = fluidContribParticles[neighParticleGlobalIdx];
                            float neighPressure = fluidContribPressure[neighParticleGlobalIdx];
                            float neighDensity = fluidContribDensity[neighParticleGlobalIdx];

//                            float pressOverDens = (fabs(neighDensity)<FLT_EPSILON ? 0.0f: (thisPressure + neighPressure) / (2.0f* neighDensity));

//                            accPressureForce = accPressureForce + (neighMass * pressOverDens * SpikyKernelGradientV_Kernel(thisParticle, neighParticle, smoothingLength));

//                            accPressureForce = accPressureForce + (neighMass * (thisPressure+neighPressure) / (neighDensity + neighDensity) * SpikyKernelGradientV_Kernel(thisParticle, neighParticle, smoothingLength));


                            accPressureForce = accPressureForce + ( ((thisPressure/(thisDensity*thisDensity)) + (neighPressure/(neighDensity*neighDensity))) * SpikyKernelGradientV_Kernel(thisParticle, neighParticle, smoothingLength) );
                        }
                    }
                }
            }
        }


        if(!accumulate)
        {
//            pressureForce[thisParticleGlobalIdx] = -1.0f * accPressureForce;
            pressureForce[thisParticleGlobalIdx] = -1.0f * thisMass* thisMass * accPressureForce;
        }
        else
        {
            pressureForce[thisParticleGlobalIdx] = pressureForce[thisParticleGlobalIdx] + (-1.0f *thisMass * thisMass * accPressureForce);
//            pressureForce[thisParticleGlobalIdx] = pressureForce[thisParticleGlobalIdx] + (-1.0f * accPressureForce);
        }
    }
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputePressureForceFluidRigid_kernel(float3 *pressureForce,
                                                                      const float *pressure,
                                                                      const float *density,
                                                                      const float mass,
                                                                      const float3 *particles,
                                                                      const uint *cellOcc,
                                                                      const uint *cellPartIdx,
                                                                      const float restDensity,
                                                                      const float *rigidVolume,
                                                                      const float3 *rigidPos,
                                                                      const uint *rigidCellOcc,
                                                                      const uint *rigidCellPartIdx,
                                                                      const uint numPoints,
                                                                      const float smoothingLength,
                                                                      const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;


    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {

        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float3 accPressureForce = make_float3(0.0f, 0.0f, 0.0f);


        float thisDensity = density[thisParticleGlobalIdx];
        float thisPressure = pressure[thisParticleGlobalIdx];
        float thisMass = mass;
        float3 thisParticle = particles[thisParticleGlobalIdx];

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = rigidCellOcc[neighCellIdx];
                    neighCellPartIdx = rigidCellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        float3 neighParticle = rigidPos[neighParticleGlobalIdx];
                        float neighVolume = rigidVolume[neighParticleGlobalIdx];
                        float pressOverDens = (fabs(thisDensity)<FLT_EPSILON ? 0.0f: (thisPressure) / (thisDensity*thisDensity));

                        accPressureForce = accPressureForce + (thisMass * neighVolume * restDensity * pressOverDens * SpikyKernelGradientV_Kernel(thisParticle, neighParticle, smoothingLength));
                    }
                }
            }
        }


        if(!accumulate)
        {
            pressureForce[thisParticleGlobalIdx] = -1.0 * accPressureForce;
//            pressureForce[thisParticleGlobalIdx] = -1.0 * thisMass * thisMass * accPressureForce;
        }
        else
        {
            pressureForce[thisParticleGlobalIdx] = pressureForce[thisParticleGlobalIdx] + (-1.0f * accPressureForce);
//            pressureForce[thisParticleGlobalIdx] = pressureForce[thisParticleGlobalIdx] + (-1.0f * thisMass * thisMass * accPressureForce);
        }
    }
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputeViscousForce_kernel(float3 *viscForce,
                                                           const float viscCoeff,
                                                           const float3 *velocity,
                                                           const float *density,
                                                           const float mass,
                                                           const float3 *position,
                                                           const uint *cellOcc,
                                                           const uint *cellPartIdx,
                                                           const uint numPoints,
                                                           const float smoothingLength)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;


    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;
        float3 accViscForce = make_float3(0.0f, 0.0f, 0.0f);


        float3 thisPos = position[thisParticleGlobalIdx];
        float3 thisVel = velocity[thisParticleGlobalIdx];
        float neighMass = mass;

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        if(neighParticleGlobalIdx == thisParticleGlobalIdx){continue;}

                        float3 neighPos = position[neighParticleGlobalIdx];
                        float3 neighVel = velocity[neighParticleGlobalIdx];
                        float neighDensity = density[neighParticleGlobalIdx];


                        float neighMassOverDen = ( (fabs(neighDensity)<FLT_EPSILON) ? 0.0f : neighMass / neighDensity );

                        accViscForce = accViscForce + ( neighMassOverDen * (neighVel - thisVel) * Poly6Laplacian_Kernel(length(thisPos - neighPos), smoothingLength) );
                    }
                }
            }
        }

        viscForce[thisParticleGlobalIdx] = -1.0f * viscCoeff * accViscForce;
    }
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputeSurfaceTensionForce_kernel(float3 *surfaceTensionForce,
                                                                  const float surfaceTension,
                                                                  const float surfaceThreshold,
                                                                  const float *density,
                                                                  const float mass,
                                                                  const float3 *position,
                                                                  const uint *cellOcc,
                                                                  const uint *cellPartIdx,
                                                                  const uint numPoints,
                                                                  const float smoothingLength)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;


    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;


        float3 thisPos = position[thisParticleGlobalIdx];
        float3 accColourFieldGrad = make_float3(0.0f, 0.0f, 0.0f);
        float accCurvature = 0.0f;
        float neighMass = mass;

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        if(neighParticleGlobalIdx == thisParticleGlobalIdx){continue;}

                        float3 neighPos = position[neighParticleGlobalIdx];
                        float neighDensity = density[neighParticleGlobalIdx];

                        float neighMassOverDen = ( (fabs(neighDensity)<FLT_EPSILON) ? 0.0f : neighMass / neighDensity );

                        accColourFieldGrad = accColourFieldGrad + ( neighMassOverDen * SpikyKernelGradientV_Kernel(thisPos, neighPos, smoothingLength) );
                        accCurvature = accCurvature + (neighMassOverDen * -Poly6Laplacian_Kernel(length(thisPos - neighPos), smoothingLength));

                    }
                }
            }
        }

        float colourFieldGradMag = length(accColourFieldGrad);
        if( colourFieldGradMag > surfaceThreshold )
        {
            accCurvature /= colourFieldGradMag;
            surfaceTensionForce[thisParticleGlobalIdx] = (surfaceTension * accCurvature * accColourFieldGrad);
        }
        else
        {
            surfaceTensionForce[thisParticleGlobalIdx] = make_float3(0.0f, 0.0f, 0.0f);
        }
    }
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputeForce_kernel(/*float3 *force,*/
                                                    float3 *pressureForce,
                                                    float3 *viscForce,
                                                    float3 *surfaceTensionForce,
//                                                    const float3 gravity,
                                                    const float viscCoeff,
                                                    const float surfaceTension,
                                                    const float surfaceThreshold,
                                                    const float *pressure,
                                                    const float *density,
                                                    const float mass,
                                                    const float3 *particles,
                                                    const float3 *velocity,
                                                    const uint *cellOcc,
                                                    const uint *cellPartIdx,
                                                    const uint numPoints,
                                                    const float smoothingLength,
                                                    const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;
    int thisCellOcc = cellOcc[thisCellIdx];


    if(!(thisParticleGlobalIdx < numPoints && threadIdx.x < thisCellOcc && thisCellIdx < gridDim.x * gridDim.y * gridDim.z))
    {
        return;
    }
//        const int num = 1024;
//        __shared__ float3 s_pos[num];
//        __shared__ float3 s_vel[num];
//        __shared__ float s_pres[num];
//        __shared__ float s_den[num];
//        __shared__ float s_mass[num];

        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;
        int neighLocalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

//        if(threadIdx.x < 27)
//        {

//            int dx = threadIdx.x % 3;
//            int dy = ((threadIdx.x - dx) / 3) % 3;
//            int dz = ((threadIdx.x - dx) - (dy*3)) / 9;

//            neighCellIdx = (blockIdx.x + dx) + ((blockIdx.y + dy) * gridDim.x) + ((blockIdx.z + dz) * gridDim.x * gridDim.y);
//            neighCellOcc = cellOcc[neighCellIdx];
//            neighCellPartIdx = cellPartIdx[neighCellIdx];

//            int scatterAddr = 0;
//            int idx = 0;
//            for(z = zMin; z <= zMax; z++)
//            {
//                for(y = yMin; y <= yMax; y++)
//                {
//                    for(x = xMin; x <= xMax; x++)
//                    {
//                        if(neighCellIdx >= (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y))
//                            continue;
//                        scatterAddr += cellOcc[neighCellIdx];
//                    }
//                }
//            }


//                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
//                    {
//                        if(scatterAddr >= num)
//                            continue;

//                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
//                        s_pos[scatterAddr] = particles[neighParticleGlobalIdx];
//                        s_vel[scatterAddr] = velocity[neighParticleGlobalIdx];
//                        s_den[scatterAddr] = density[neighParticleGlobalIdx];
//                        s_pres[scatterAddr] = pressure[neighParticleGlobalIdx];
//                        s_mass[scatterAddr] = mass[neighParticleGlobalIdx];
//                        scatterAddr++;

//                    }
//        }

//        __syncthreads();






        float thisPressure = pressure[thisParticleGlobalIdx];
        float3 thisPos = particles[thisParticleGlobalIdx];
        float3 thisVel = velocity[thisParticleGlobalIdx];
        float3 accPressureForce = make_float3(0.0f, 0.0f, 0.0f);
        float3 accViscForce = make_float3(0.0f, 0.0f, 0.0f);
        float3 accColourFieldGrad = make_float3(0.0f, 0.0f, 0.0f);
        float accCurvature = 0.0f;
        float neighMass = mass;

        int idx = 0;
        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = cellOcc[neighCellIdx];
                    neighCellPartIdx = cellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        if(neighParticleGlobalIdx != thisParticleGlobalIdx)
                        {
                            float3 neighPos = particles[neighParticleGlobalIdx];
                            float3 neighVel = velocity[neighParticleGlobalIdx];
                            float neighPressure = pressure[neighParticleGlobalIdx];
                            float neighDensity = density[neighParticleGlobalIdx];

                            float3 gradW = SpikyKernelGradientV_Kernel(thisPos, neighPos, smoothingLength);
                            float W = Poly6Laplacian_Kernel(length(thisPos - neighPos), smoothingLength);

                            float pressOverDens = (fabs(neighDensity)<FLT_EPSILON ? 0.0f: (thisPressure + neighPressure) / (2.0f* neighDensity));
                            accPressureForce = accPressureForce + (neighMass * pressOverDens * gradW);

                            float neighMassOverDen = ( (fabs(neighDensity)<FLT_EPSILON) ? 0.0f : neighMass / neighDensity );
                            accViscForce = accViscForce + ( neighMassOverDen * (neighVel - thisVel) * W );

                            accColourFieldGrad = accColourFieldGrad + ( neighMassOverDen * gradW );
                            accCurvature = accCurvature + (neighMassOverDen * -W);
                        }
                        else
                        {
                            idx++;
                        }
                    }
                }
            }
        }


        accPressureForce = -1.0f * accPressureForce;
        accPressureForce = accumulate ? pressureForce[thisParticleGlobalIdx] + accPressureForce : accPressureForce;
        pressureForce[thisParticleGlobalIdx] = accPressureForce;


        accViscForce = -1.0f * viscCoeff * accViscForce;
        viscForce[thisParticleGlobalIdx] = accViscForce;


        float colourFieldGradMag = length(accColourFieldGrad);
        float3 accSurfTenForce = (colourFieldGradMag > surfaceThreshold ) ? (-1.0f * surfaceTension * (accCurvature/colourFieldGradMag) * accColourFieldGrad) : make_float3(0.0f,0.0f,0.0f);
        surfaceTensionForce[thisParticleGlobalIdx] = accSurfTenForce;





//        // re-initialise forces to zero
//        float3 accForce = make_float3(0.0f, 0.0f, 0.0f);

//        // Add external force
//        float3 extForce = externalForce[thisCellIdx];
//        if(isnan(extForce.x) || isnan(extForce.y) || isnan(extForce.z))
//        {
//            printf("nan external force\n");
//        }
//        else
//        {
//            accForce = accForce + extForce;
//        }


//        // Add pressure force
//        if(isnan(accPressureForce.x) || isnan(accPressureForce.y) || isnan(accPressureForce.z))
//        {
//            printf("nan pressure force\n");
//        }
//        else
//        {
//            accForce = accForce + accPressureForce;
//        }

//        // Add Viscous force
//        if(isnan(accViscForce.x) || isnan(accViscForce.y) || isnan(accViscForce.z))
//        {
//            printf("nan visc force\n");
//        }
//        else
//        {
//            accForce = accForce + accViscForce;
//        }

//        // Add surface tension force
//        if(isnan(accSurfTenForce.x) || isnan(accSurfTenForce.y) || isnan(accSurfTenForce.z))
//        {
//            printf("nan surfTen force\n");
//        }
//        else
//        {
//            //printf("%f, %f, %f\n",surfTenForce.x, surfTenForce.y,surfTenForce.z);
//            accForce = accForce + accSurfTenForce;
//        }


//        // Work out acceleration from force
//        float3 acceleration = accForce / mass[thisParticleGlobalIdx];

//        // Add gravity acceleration
//        acceleration = acceleration + gravity;

//        // Set particle force
//        force[thisParticleGlobalIdx] = acceleration;

}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::ComputeTotalForce_kernel(const bool accumulatePressure,
                                                         const bool accumulateViscous,
                                                         const bool accumulateSurfTen,
                                                         const bool accumulateExternal,
                                                         const bool accumulateGravity,
                                                         float3 *force,
                                                         const float3 *externalForce,
                                                         const float3 *pressureForce,
                                                         const float3 *viscousForce,
                                                         const float3 *surfaceTensionForce,
                                                         const float3 gravity,
                                                         const float mass,
                                                         const float3 *particles,
                                                         const float3 *velocities,
                                                         const uint *cellOcc,
                                                         const uint *cellPartIdx,
                                                         const uint numPoints,
                                                         const float smoothingLength)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;

    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        // re-initialise forces to zero
        float3 accForce = make_float3(0.0f, 0.0f, 0.0f);

        // Add external force
        if(accumulateExternal)
        {
            float3 extForce = externalForce[thisCellIdx];
            if(isnan(extForce.x) || isnan(extForce.y) || isnan(extForce.z))
            {
                printf("nan external force\n");
            }
            else
            {
                accForce = accForce + extForce;
            }
        }


        // Add pressure force
        if(accumulatePressure)
        {
            float3 pressForce = pressureForce[thisParticleGlobalIdx];
            if(isnan(pressForce.x) || isnan(pressForce.y) || isnan(pressForce.z))
            {
                printf("nan pressure force\n");
            }
            else
            {
                accForce = accForce + pressForce;
            }
        }


        // Add Viscous force
        if(accumulateViscous)
        {
            float3 viscForce = viscousForce[thisParticleGlobalIdx];
            if(isnan(viscForce.x) || isnan(viscForce.y) || isnan(viscForce.z))
            {
                printf("nan visc force\n");
            }
            else
            {
                accForce = accForce + viscForce;
            }
        }


        // Add surface tension force
        if(accumulateSurfTen)
        {
            float3 surfTenForce = surfaceTensionForce[thisParticleGlobalIdx];
            if(isnan(surfTenForce.x) || isnan(surfTenForce.y) || isnan(surfTenForce.z))
            {
                printf("nan surfTen force\n");
            }
            else
            {
                //printf("%f, %f, %f\n",surfTenForce.x, surfTenForce.y,surfTenForce.z);
                accForce = accForce + surfTenForce;
            }
        }


        // Work out acceleration from force
        float3 acceleration = accForce / mass;

        // Add gravity acceleration
        if(accumulateGravity)
        {
            acceleration = acceleration + gravity;
        }

        // Set particle force
        force[thisParticleGlobalIdx] = acceleration;
    }
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::Integrate_kernel(float3 *force,
                                                 float3 *particles,
                                                 float3 *velocities,
                                                 const float _dt,
                                                 const uint numPoints)
{
    uint idx = threadIdx.x + (blockIdx.x * blockDim.x);

    if(idx < numPoints)
    {
        //---------------------------------------------------------
        // Good old instable Euler integration - ONLY FOR TESTING
        float3 oldPos = particles[idx];
        float3 oldVel = velocities[idx];

        float3 newVel = oldVel + (_dt * force[idx]);
        float3 newPos = oldPos + (_dt * newVel);

        //---------------------------------------------------------
        // Verlet/Leapfrog integration
//        float3 newPos = oldPos + (oldVel * _dt) + (0.5f * force[idx] * _dt * _dt);
//        float3 newVel = oldVel + (0.5 * (force[idx] + force[idx]) * _dt);

        //---------------------------------------------------------
        // TODO:
        // Verlet integration
        // RK4 integration

        //---------------------------------------------------------
        // Error checking and setting new values

        if(isnan(newVel.x) || isnan(newVel.y) || isnan(newVel.z))
        {
            printf("nan vel\n");
        }
        else
        {
            velocities[idx] = newVel;
        }

        if(isnan(newPos.x) || isnan(newPos.y) || isnan(newPos.z))
        {
            printf("nan pos\n");
        }
        else
        {
            particles[idx] = newPos;
        }
    }
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::HandleBoundaries_Kernel(float3 *particles,
                                                        float3 *velocities,
                                                        const float boundary,
                                                        const uint numPoints)
{
    uint idx = threadIdx.x + (blockIdx.x * blockDim.x);

    if(idx < numPoints)
    {

        float3 pos = particles[idx];
        float3 vel = velocities[idx];

        float boundaryDamper = 0.4f;

        if(pos.x < -boundary)
        {
           pos.x = -boundary  + fabs(fabs(pos.x) - boundary);
           vel = make_float3(boundaryDamper*fabs(vel.x),vel.y,vel.z);
        }
        if(pos.x > boundary)
        {
           pos.x = boundary - fabs(fabs(pos.x) - boundary);
           vel = make_float3(-boundaryDamper*fabs(vel.x),vel.y,vel.z);
        }

        if(pos.y < -boundary)
        {
           pos.y = -boundary + fabs(fabs(pos.y) - boundary);
           vel = make_float3(vel.x,boundaryDamper*fabs(vel.y),vel.z);
        }
        if(pos.y > boundary)
        {
           pos.y = boundary - fabs(fabs(pos.y) - boundary);
           vel = make_float3(vel.x,-boundaryDamper*fabs(vel.y),vel.z);
        }

        if(pos.z < -boundary)
        {
           pos.z = -boundary + fabs(fabs(pos.z) - boundary);
           vel = make_float3(vel.x,vel.y,boundaryDamper*fabs(vel.z));
        }
        if(pos.z > boundary)
        {
           pos.z = boundary - fabs(fabs(pos.z) - boundary);
           vel = make_float3(vel.x,vel.y,-boundaryDamper*fabs(vel.z));
        }

        particles[idx] = pos;
        velocities[idx] = vel;
    }
}

//--------------------------------------------------------------------------------------------------------------------

__global__ void sphGPU_Kernels::InitParticleAsCube_Kernel(float3 *particles,
                                                          float3 *velocities,
                                                          float *densities,
                                                          const float restDensity,
                                                          const uint numParticles,
                                                          const uint numPartsPerAxis,
                                                          const float scale)
{

    uint x = threadIdx.x + (blockIdx.x * blockDim.x);
    uint y = threadIdx.y + (blockIdx.y * blockDim.y);
    uint z = threadIdx.z + (blockIdx.z * blockDim.z);
    uint idx = x + (y * numPartsPerAxis) + (z * numPartsPerAxis * numPartsPerAxis);

    if(x >= numPartsPerAxis || y >= numPartsPerAxis || z >= numPartsPerAxis || idx >= numParticles)
    {
        return;
    }

    float posX = scale * (x - (0.5f * numPartsPerAxis));
    float posY = scale * (y - (0.5f * numPartsPerAxis));
    float posZ = scale * (z - (0.5f * numPartsPerAxis));

    particles[idx] = make_float3(posX, posY, posZ);
    velocities[idx] = make_float3(0.0f, 0.0f, 0.0f);
    densities[idx] = restDensity;
}



//--------------------------------------------------------------------------------------------------------------------
// Algae functions
__global__ void sphGPU_Kernels::ComputeAdvectionForce(float3 *pos,
                                                      float3 *vel,
                                                      float3 *advectForce,
                                                      const uint *cellOcc,
                                                      const uint *cellPartIdx,
                                                      const float3 *advectorPos,
                                                      const float3 *advectorForce, const float *advectorDensity, const float advectorMass,
                                                      const uint *advectorCellOcc,
                                                      const uint *advectorCellPartIdx,
                                                      const uint numPoints,
                                                      const float smoothingLength,
                                                      const bool accumulate)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;


    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;


        float3 thisPos = pos[thisParticleGlobalIdx];
        float3 accForce = make_float3(0.0f, 0.0f, 0.0f);
//        vel[thisParticleGlobalIdx] = vel[thisParticleGlobalIdx]*0.9f;//make_float3(0.0f, 0.0f, 0.0f);

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = advectorCellOcc[neighCellIdx];
                    neighCellPartIdx = advectorCellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        float3 neighPos = advectorPos[neighParticleGlobalIdx];

                        float W = Poly6Kernel_Kernel(length(thisPos-neighPos), smoothingLength);
                        float invDensity = 1.0f / advectorDensity[neighParticleGlobalIdx];

                        accForce = accForce + (advectorForce[neighParticleGlobalIdx] * W * invDensity);

//                        accForce = accForce + ((neighPos - thisPos) *0.1f* W);
                    }
                }
            }
        }

        accForce = (accForce * advectorMass * 1.00f);// + make_float3(0.0f, -0.8f, 0.0f);

        advectForce[thisParticleGlobalIdx] = accForce;
    }
}

//--------------------------------------------------------------------------------------------------------------------
__global__ void sphGPU_Kernels::AdvectParticle(float3 *pos,
                                               float3 *vel,
                                               const uint *cellOcc,
                                               const uint *cellPartIdx,
                                               const float3 *advectorPos,
                                               const float3 *advectorVel,
                                               const float *advectorDensity,
                                               const float advectorMass,
                                               const uint *advectorCellOcc,
                                               const uint *advectorCellPartIdx,
                                               const uint numPoints,
                                               const float smoothingLength,
                                               const float deltaTime)
{
    int thisCellIdx = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * gridDim.x * gridDim.y);
    int thisParticleGlobalIdx = cellPartIdx[thisCellIdx] + threadIdx.x;


    if(thisParticleGlobalIdx < numPoints && threadIdx.x < cellOcc[thisCellIdx] && thisCellIdx < gridDim.x * gridDim.y * gridDim.z)
    {
        int neighCellIdx;
        int neighCellOcc;
        int neighCellPartIdx;
        int neighParticleGlobalIdx;

        int x, y, z;
        int xMin = ((blockIdx.x==0)?0:-1);
        int yMin = ((blockIdx.y==0)?0:-1);
        int zMin = ((blockIdx.z==0)?0:-1);
        int xMax = ((blockIdx.x==gridDim.x-1)?0:1);
        int yMax = ((blockIdx.y==gridDim.y-1)?0:1);
        int zMax = ((blockIdx.z==gridDim.z-1)?0:1);

        int neighLocalIdx;


        float3 thisPos = pos[thisParticleGlobalIdx];
        float3 accVel = make_float3(0.0f, 0.0f, 0.0f);

        for(z = zMin; z <= zMax; z++)
        {
            for(y = yMin; y <= yMax; y++)
            {
                for(x = xMin; x <= xMax; x++)
                {

                    neighCellIdx = (blockIdx.x + x) + ((blockIdx.y + y) * gridDim.x) + ((blockIdx.z + z) * gridDim.x * gridDim.y);
                    neighCellOcc = advectorCellOcc[neighCellIdx];
                    neighCellPartIdx = advectorCellPartIdx[neighCellIdx];

                    for(neighLocalIdx=0; neighLocalIdx<neighCellOcc; neighLocalIdx++)
                    {
                        neighParticleGlobalIdx = neighCellPartIdx + neighLocalIdx;
                        float3 neighPos = advectorPos[neighParticleGlobalIdx];
                        float3 neighVel = advectorVel[neighParticleGlobalIdx];

                        float W = Poly6Kernel_Kernel(length(thisPos-neighPos), smoothingLength);
                        float invDensity = 1.0f / advectorDensity[neighParticleGlobalIdx];
                        accVel = accVel + (neighVel * W * invDensity);
//                        accVel = accVel + ((neighPos - thisPos) *0.1f* W);
                    }
                }
            }
        }

        vel[thisParticleGlobalIdx] = (vel[thisParticleGlobalIdx]*0.5f) + (accVel * advectorMass * 0.50f);
        pos[thisParticleGlobalIdx] = thisPos + (accVel * deltaTime);
    }
}

//--------------------------------------------------------------------------------------------------------------------
__global__ void sphGPU_Kernels::ComputeBioluminescence(const float *pressure,
                                                       float *prevPressure,
                                                       float *illumination,
                                                       const uint numPoints)
{
    uint idx = threadIdx.x + (blockIdx.x * blockDim.x);

    if(idx < numPoints)
    {
        float currIllum = illumination[idx];
        float beta = 0.01f;
        float press = pressure[idx];
        float prevPress = prevPressure[idx];
        prevPressure[idx] = press;
        float deltaPress = fabs(press - prevPress);

        float deltaIllum = (deltaPress > beta) ? 0.001 : -0.0001f;

        currIllum += deltaIllum;
        currIllum = (currIllum < 0.0f) ? 0.0f : currIllum;
        currIllum = (currIllum > 0.02f) ? 0.02f : currIllum;

        illumination[idx] = 0.04f;//currIllum;

    }

}
