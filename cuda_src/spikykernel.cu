#include "hip/hip_runtime.h"
#include "../cuda_inc/spikykernel.cuh"

//SpikyKernel::SpikyKernel(float _h) : SmoothKernel()
//{
//    m_h = _h;
//}

//SpikyKernel::~SpikyKernel()
//{

//}

//__host__ __device__ float SpikyKernel::Eval(float _x)
//{
//    if(fabs(_x) > m_h || _x < 0.0f)
//    {
//        return 0.0f;
//    }
//    else
//    {
//        return (15.0f/(HIP_PI_F * pow(m_h, 6.0f))) * pow((m_h-_x), 3.0f);
//    }
//}

//__host__ __device__ float SpikyKernel::Grad(float _x)
//{
//    if(fabs(_x) > m_h || fabs(_x) <= FLT_EPSILON)
//    {
//        return 0.0f;
//    }
//    else
//    {
//        float coeff = - (45.0f/(HIP_PI_F*pow(m_h,6.0f)));
//        return coeff * pow((m_h-_x), 2.0f);
//    }
//}

//__host__ __device__ float3 SpikyKernel::Grad(float3 _x)
//{
//    float distance = length(_x);
//    if(fabs(distance) <= FLT_EPSILON)
//    {
//        return make_float3(0.f, 0.f, 0.f);
//    }
//    else
//    {
//        float c = Grad(distance);

//        return (c * _x/distance);
//    }
//}

//__host__ __device__ float SpikyKernel::Laplace(float _x)
//{
//    return 0.0f;
//}

